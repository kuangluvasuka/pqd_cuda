#include "hip/hip_runtime.h"
#include "pqd1.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


/* GPU kernel prototypes */
__global__ void gpu_pot_prop(double* psi, double* u);
__global__ void gpu_kin_prop(double* psi, double* wrk, double* al, double* blx, double* bux, int t);

extern "C" void gpu_init(int myid) {
    int dev_num;
	hipSetDevice(myid % 2);
    hipGetDevice(&dev_num);
    printf("myid is %d, GPU id is %d\n", myid, dev_num);
    
    hipMalloc((void**) &dev_psi, sizeof(double) * 2 * (NX+2));
    hipMalloc((void**) &dev_wrk, sizeof(double) * 2 * (NX+2));
    hipMalloc((void**) &dev_u, sizeof(double) * 2 * (NX+2));
    hipMalloc((void**) &dev_al, sizeof(double) * 2 * 2);
    hipMalloc((void**) &dev_blx, sizeof(double) * 2 * (NX+2) * 2);
    hipMalloc((void**) &dev_bux, sizeof(double) * 2 * (NX+2) * 2);
    
    hipMemcpy2D(dev_u, 2*sizeof(double), u, 2*sizeof(double), 2*sizeof(double), NX+2, hipMemcpyHostToDevice);
    hipMemcpy2D(dev_al, 2*sizeof(double), al, 2*sizeof(double), 2*sizeof(double), 2, hipMemcpyHostToDevice);
    hipMemcpy(dev_blx, blx, 2*(NX+2)*2*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_bux, bux, 2*(NX+2)*2*sizeof(double), hipMemcpyHostToDevice);
}



extern "C" void gpu_lanch_pot_prop() {
    hipMemcpy2D(dev_psi, 2*sizeof(double), psi, 2*sizeof(double), 2*sizeof(double), NX+2, hipMemcpyHostToDevice);
    
    gpu_pot_prop<<<1, NX>>>(dev_psi, dev_u);
    
    hipMemcpy2D(psi, 2*sizeof(double), dev_psi, 2*sizeof(double), 2*sizeof(double), NX+2, hipMemcpyDeviceToHost);
}
    
extern "C" void gpu_lanch_kin_prop(int t) {
    hipMemcpy2D(dev_psi, 2*sizeof(double), psi, 2*sizeof(double), 2*sizeof(double), NX+2, hipMemcpyHostToDevice);
    hipMemcpy2D(dev_wrk, 2*sizeof(double), wrk, 2*sizeof(double), 2*sizeof(double), NX+2, hipMemcpyHostToDevice);
    
    gpu_kin_prop<<<1, NX>>>(dev_psi, dev_wrk, dev_al, dev_blx, dev_bux, t);

    hipMemcpy2D(wrk, 2*sizeof(double), dev_wrk, 2*sizeof(double), 2*sizeof(double), NX+2, hipMemcpyDeviceToHost);

}

__global__ void gpu_pot_prop(double* psi, double* u)
{
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    int sx = tid + 1;
    double wr, wi;
    if (sx <= NX) {
        double* row_psi = (double*)((char*)psi + sx * 2 * sizeof(double));
        double* row_u = (double*)((char*)u + sx * 2 * sizeof(double));

        wr = row_u[0]*row_psi[0]-row_u[1]*row_psi[1];
		wi = row_u[0]*row_psi[1]+row_u[1]*row_psi[0];
		row_psi[0] = wr;
		row_psi[1] = wi;

    }

}

__global__ void gpu_kin_prop(double* psi, double* wrk, double* al, double* blx, double* bux, int t)
{
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    int sx = tid + 1;
	double wr,wi;
    if (sx <= NX) {
        double* row_blx = (double*)((char*)blx + t * NX * 2 * sizeof(double) + sx * 2 * sizeof(double));
        double* row_bux = (double*)((char*)bux + t * NX * 2 * sizeof(double) + sx * 2 * sizeof(double));
        double* row_psi = (double*)((char*)psi + sx * 2 * sizeof(double));
        double* row_psi_r = (double*)((char*)psi + (sx+1) * 2 * sizeof(double));
        double* row_psi_l = (double*)((char*)psi + (sx-1) * 2 * sizeof(double));
        double* row_al = (double*)((char*)al + t * 2 * sizeof(double));
        double* row_wrk = (double*)((char*)wrk + sx * 2 * sizeof(double)); 
        
        wr = row_al[0]*row_psi[0]-row_al[1]*row_psi[1];
        wi = row_al[0]*row_psi[1]+row_al[1]*row_psi[0];
        wr += (row_blx[0]*row_psi_l[0]-row_blx[1]*row_psi_l[1]);
        wi += (row_blx[0]*row_psi_l[1]+row_blx[1]*row_psi_l[0]);
        wr += (row_bux[0]*row_psi_r[0]-row_bux[1]*row_psi_r[1]);
        wi += (row_bux[0]*row_psi_r[1]+row_bux[1]*row_psi_r[0]);
        row_wrk[0] = wr;
        row_wrk[1] = wi;
    }

}





